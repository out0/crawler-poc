
#include <hip/hip_runtime.h>

#define NUM_THREADS 32
#define TILE_SIZE 1024 // NUM_THREADS * NUM_THREADS

typedef unsigned char uchar;

template <typename T>
__global__ void cudaMatrixMul(T *a, T *b, T *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col > k || row > m)
        return;

    __shared__ int tileA[TILE_SIZE];
    __shared__ int tileB[TILE_SIZE];

    T sum = 0;

    for (int i = 0; i < n; i += blockDim.x)
    {
        tileA[threadIdx.y * blockDim.x + threadIdx.x] = a[row * n + i + threadIdx.x];
        tileB[threadIdx.y * blockDim.x + threadIdx.x] = b[(i + threadIdx.y) * k + col];

        __syncthreads();

        for (int j = 0; j < blockDim.x; j++)
        {
            sum += tileA[threadIdx.y * blockDim.x + j] * tileB[j * blockDim.x + threadIdx.x];
        }

        __syncthreads();
    }

    if (row < m && col < k)
        c[row * k + col] = sum;
}

int padDimensionSize(int dim)
{
    return dim + NUM_THREADS - dim % NUM_THREADS;
}

template <typename T>
void matmul(T *m1, T *m2, T *result, int m, int n, int k)
{
    T *cudaM1;
    T *cudaM2;
    T *cudaResult;

    size_t sizeM1 = m * n * sizeof(T);
    size_t sizeM2 = n * k * sizeof(T);
    size_t sizeResult = m * k * sizeof(T);

    hipMalloc(&cudaM1, sizeM1);
    hipMalloc(&cudaM2, sizeM2);
    hipMalloc(&cudaResult, sizeResult);

    hipMemcpy(cudaM1, m1, sizeM1, hipMemcpyHostToDevice);
    hipMemcpy(cudaM2, m2, sizeM2, hipMemcpyHostToDevice);

    int BLOCKS_X = (k / NUM_THREADS);
    int BLOCKS_Y = (m / NUM_THREADS);

    if (BLOCKS_X == 0)
        BLOCKS_X = 1;

    if (BLOCKS_Y == 0)
        BLOCKS_Y = 1;

    dim3 threads(NUM_THREADS, NUM_THREADS);
    dim3 blocks(BLOCKS_X, BLOCKS_Y);

    cudaMatrixMul<<<blocks, threads>>>(cudaM1, cudaM2, cudaResult, m, n, k);

    hipMemcpy(result, cudaResult, sizeResult, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(cudaM1);
    hipFree(cudaM2);
    hipFree(cudaResult);
}

template void matmul(float *m1, float *m2, float *result, int m, int n, int k);
template void matmul(int *m1, int *m2, int *result, int m, int n, int k);
template void matmul(uchar *m1, uchar *m2, uchar *result, int m, int n, int k);
